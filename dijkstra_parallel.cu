#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scatter.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "grid.h"

extern "C"
{

int minDistance(float* dist, int* visited, int size)
{
	float min = __FLT_MAX__;

	thrust::device_vector<float> d_dist(size, min);
	
	thrust::device_vector<int> d_map(size);
	thrust::sequence(d_map.begin(), d_map.end());

	thrust::scatter_if(thrust::device, dist, dist + size, d_map.begin(), visited, d_dist.begin());

	thrust::device_vector<float>::iterator iter = thrust::min_element(d_dist.begin(), d_dist.end());
	
	int min_index = iter - d_dist.begin();
	return min_index;
}


void Grid::dijkstraShortestPath(Vertex src) {
	thrust::device_vector<float> dist(num_rows * num_columns, __FLT_MAX__);	// stores distance of each node from source
	thrust::device_vector<int> visited(num_rows * num_columns, 1);
	float* dist_p = thrust::raw_pointer_cast(&dist[0]);
	int* visited_p = thrust::raw_pointer_cast(&visited[0]);	
	

	thrust::host_vector<int> backtrack(num_rows * num_columns, -1);	// used to backtrack path from source to each node
	
	dist[src.x * num_columns + src.y] = 0;


	for (int i = 0; i < num_columns*num_rows - 1; i++)
	{
		
		int v = minDistance(dist_p, visited_p, num_rows*num_columns);
		
		visited[v] = 0;
		for (int k = adj[v]; k < adj[v+1]; k++)
		{
			if(!edges[k]->selected)
					continue;
			
			Vertex u = edges[k]->head;
			int u_index = u.x * num_columns + u.y;

			if (! (visited[u_index]==0) && dist[v] + edges[k]->weight < dist[u_index]) {
					backtrack[u_index] = v;
					dist[u_index] = edges[k]->weight + dist[v];
			}
		}
	}
	

	/*	Mark nodes along the shortest path using backtracking	*/
	int u = num_columns * num_rows - 1;
	while (u != -1) {
		selected[u] = true;
		u = backtrack[u];
	}
}

}