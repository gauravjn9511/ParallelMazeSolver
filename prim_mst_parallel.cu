#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scatter.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "grid.h"
using namespace std;

extern "C"
{

int get_closest_node(float* dist, int* included, int size)
{
	float min = __FLT_MAX__;

	thrust::device_vector<float> d_dist(size, min);

	thrust::device_vector<int> d_map(size);
	thrust::sequence(d_map.begin(), d_map.end());

	thrust::scatter_if(thrust::device, dist, dist + size, d_map.begin(), included, d_dist.begin());

	thrust::device_vector<float>::iterator iter = thrust::min_element(d_dist.begin(), d_dist.end());
	
	int min_index = iter - d_dist.begin();

	return min_index;
}

void Grid::prim_mst()
{
	int num_vertices = num_rows * num_columns;
	thrust::device_vector<float> dist(num_vertices, __FLT_MAX__);	// stores distance of each node from set
	thrust::device_vector<int> included(num_vertices, 1);

	float* dist_p = thrust::raw_pointer_cast(&dist[0]);
	int* included_p = thrust::raw_pointer_cast(&included[0]);	
	
	std::vector<int> backtrack(num_vertices, -1);	// used to backtrack path from source to each node
	dist[0] = 0;

	for (int i = 0; i < num_vertices-1; i++)
	{
		int v = get_closest_node(dist_p, included_p, num_vertices);
		included[v] = 0;

		for (int k = adj[v]; k < adj[v+1]; k++) {

			Vertex u = edges[k]->head;
			int u_index = u.x * num_columns + u.y;
			if (! (included[u_index]==0) && edges[k]->weight < dist[u_index]) {
				// backtrack[u_index] = v;
				dist[u_index] = edges[k]->weight;
				backtrack[u_index] = v;
            	// cout << "("<< u.x <<","<<u.y<<")" << " -- " << "("<< u.x <<","<<u.y<<")" << endl;
			}
		}
	}
	
	for(int u = 0; u < backtrack.size(); u++)
	{
		int v = backtrack[u];	// we must add an edge from u to v in the MST
		Vertex u1(u/num_columns, u%num_columns);
		Vertex v1(v/num_columns, v%num_columns);
		select_reverse_edge(u1, v1);
		select_reverse_edge(v1, u1);
	}
}

}